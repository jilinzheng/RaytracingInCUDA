#include "hip/hip_runtime.h"
// Note: Watch out! Include order will matter!
#include "rtweekend.h"
#include "hittable.h"
#include "color.h"
#include "camera.h"
#include "hiprand/hiprand_kernel.h"
#include "material.h"
#include <iostream>
#include <iomanip>

// assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__global__ void update_world_pointer(world *w, sphere *spheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        w->spheres = spheres;
    }
}

__global__ void update_material_pointers(sphere* d_spheres, material* d_materials, int num_spheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < num_spheres; ++i)
            d_spheres[i].mat = &d_materials[i];
    }
}

int main() {
    // select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    /* timing setup */
    hipEvent_t gpu_start, gpu_stop;
    float gpu_elapsed;

    // create cuda events
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);

    /* image/camera configuration */
    camera cam;

    // try to set dimensions divisible by warp size (32)
    // and threads per row (8)

    // match the CUDA reference
    // cam.img_width    = 1280;
    // cam.img_height   = 600;

    // match serial cpu baseline/reference
    // cam.img_width    = 640;
    // cam.img_height   = 360;

    // chapter 12
    // cam.img_width   = 712;
    // cam.img_height  = 400;

    // chapter 14 final world
    // NOTE: does not fit nicely into warp size nor block dim
    // cam.img_width   = 1200;
    // cam.img_height  = 675;

    // 720p
    cam.img_width   = 1280;
    cam.img_height  = 720;

    // samples to take around a pixel for antialiasing
    cam.samples_per_pixel = 10;

    // maximum recursion depth (implemented with for-loop)
    cam.max_depth = 20;

    // positonable camera
    cam.vfov        = 20;
    cam.lookfrom    = point3(13,2,3);
    cam.lookat      = point3(0,0,0);
    cam.vup         = vec3(0,1,0);

    // defocus blur
    cam.defocus_angle = 0.6;
    cam.focus_dist    = 10.0;

    // initialize the camera given the above parameters
    cam.initialize();
    /* end image/camera configuration*/


    // total pixels
    int num_pixels = cam.img_width * cam.img_height;
    // buffer to store device-calculated pixels, to later be printed on host;
    // using Unified Memory, i.e., managed system accessible by both host and device
    // underlying implementation goes onto device global memory
    vec3 *pixel_buffer;
    CUDA_SAFE_CALL(hipMallocManaged((void **)&pixel_buffer, num_pixels*sizeof(vec3)));

    // square blocks to start
    int num_threads_per_block_row = 8;
    dim3 dimGrid(cam.img_width/num_threads_per_block_row,
        cam.img_height/num_threads_per_block_row);
    dim3 dimBlock(num_threads_per_block_row,num_threads_per_block_row);


    /* world creation */
    // host allocations and initializations
    // chapter 14 final world
    // 1 ground, 22*22 small spheres, 3 big spheres 
    int num_materials = 1+22*22+3;
    int num_spheres = num_materials;

    material *h_materials = new material[num_materials];
    sphere *h_spheres = new sphere[num_spheres];

    // ground sphere
    h_materials[0] = material(MaterialType::LAMBERTIAN, color(0.5,0.5,0.5));
    h_spheres[0] = sphere(point3(0,-1000,0), 1000, &h_materials[0]);

    // small spheres
    for (int a = -11; a < 11; ++a) {
        for (int b = -11; b < 11; ++b) {
            double choose_mat = random_double();
            point3 center(a+0.9*random_double(), 0.2, b+0.9*random_double());

            if ((center - point3(4,0.2,0)).length() > 0.9) {
                // scale i to start from 1 and index sequentially
                // zero-based a * total b values + zero-based b + 1
                // 1 is for the already-created ground sphere
                int i = (a+11) * 22 + (b+11) + 1;

                // diffuse
                if (choose_mat < 0.8) {
                    color albedo    = color::random() * color::random();
                    h_materials[i]  = material(MaterialType::LAMBERTIAN, albedo);
                    h_spheres[i]    = sphere(center, 0.2, &h_materials[i]);
                }
                // metal
                else if (choose_mat < 0.95) {
                    color albedo    = color::random(0.5,1.0);
                    double fuzz      = random_double(0.0,0.5);
                    h_materials[i]  = material(MaterialType::METAL, albedo, fuzz);
                    h_spheres[i]    = sphere(center, 0.2, &h_materials[i]);
                }
                // glass
                else {
                    h_materials[i]  = material(MaterialType::DIELETRIC, 1.5);
                    h_spheres[i]    = sphere(center, 0.2, &h_materials[i]);
                }
            }
        }
    }

    // big spheres, start index after ground and small spheres
    int i = 1+22*22;
    // middle sphere
    h_materials[i] = material(MaterialType::DIELETRIC, 1.5);
    h_spheres[i]   = sphere(point3(0,1,0), 1.0, &h_materials[i]);
    // rear sphere
    h_materials[i+1] = material(MaterialType::LAMBERTIAN, color(0.4,0.2,0.1));
    h_spheres[i+1]   = sphere(point3(-4,1,0), 1.0, &h_materials[i+1]);
    // front sphere
    h_materials[i+2] = material(MaterialType::METAL, color(0.7,0.6,0.5), 0.0);
    h_spheres[i+2]   = sphere(point3(4,1,0), 1.0, &h_materials[i+2]);

    world *h_world = new world(h_spheres,num_spheres);

    // device allocations and transfers
    material *d_materials;
    CUDA_SAFE_CALL(hipMalloc(&d_materials,num_materials*sizeof(material)));
    CUDA_SAFE_CALL(hipMemcpy(d_materials,h_materials,num_materials*sizeof(material),
        hipMemcpyHostToDevice));

    sphere *d_spheres;
    CUDA_SAFE_CALL(hipMalloc(&d_spheres, num_spheres*sizeof(sphere)));
    CUDA_SAFE_CALL(hipMemcpy(d_spheres,h_spheres,num_spheres*sizeof(sphere),
        hipMemcpyHostToDevice));

    world *d_world;
    CUDA_SAFE_CALL(hipMalloc(&d_world,sizeof(world)));
    CUDA_SAFE_CALL(hipMemcpy(d_world,h_world,sizeof(world),
        hipMemcpyHostToDevice));

    // update world and material pointers since host pointers are invalid after transfer
    // after transferring to device
    update_world_pointer<<<1,1>>>(d_world, d_spheres);
    update_material_pointers<<<1,1>>>(d_spheres, d_materials, num_spheres);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    /* end world creation*/


    // setup random number generation in device
    hiprandState *d_rand_states;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_rand_states, num_pixels*sizeof(hiprandState)));
    init_rng<<<dimGrid, dimBlock>>>(cam.img_width, cam.img_height, d_rand_states);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // call the render() kernel
    // record event on default stream
    hipEventRecord(gpu_start, 0);
    render<<<dimGrid, dimBlock>>>(pixel_buffer, cam, d_world, d_rand_states);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    // stop and destroy timer
    hipEventRecord(gpu_stop,0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
    std::clog << std::fixed << std::setprecision(8) << std::setw(15) << gpu_elapsed << std::endl;
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    // output pixel_buffer as a .ppm image
    const interval intensity(0.000,0.999);
    std::cout << "P3\n" << cam.img_width << " " << cam.img_height << "\n255\n";
    for (int j = 0; j < cam.img_height; ++j) {      // rows
        for (int i = 0; i < cam.img_width; ++i) {   // cols
            size_t pixel_index = j * cam.img_width + i;
            vec3 pixel = pixel_buffer[pixel_index];
            int r = int(256 * intensity.clamp(pixel.x()));
            int g = int(256 * intensity.clamp(pixel.y()));
            int b = int(256 * intensity.clamp(pixel.z()));
            std::cout << r << " " << g << " " << b << "\n";
        }
    }

    // hipFree device allocations, delete host heap allocations
    CUDA_SAFE_CALL(hipFree(pixel_buffer));
    CUDA_SAFE_CALL(hipFree(d_materials));
    CUDA_SAFE_CALL(hipFree(d_spheres));
    CUDA_SAFE_CALL(hipFree(d_world));
    CUDA_SAFE_CALL(hipFree(d_rand_states));
    delete[] h_materials;
    delete[] h_spheres;
    delete h_world;

    return 0;
}
