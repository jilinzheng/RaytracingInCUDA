#include "hip/hip_runtime.h"

//==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

// Note: Watch out! Include order will matter!
#include "rtweekend.h"
// #include "camera.h"
// #include "hittable.h"
// #include "hittable_list.h"
// #include "material.h"
// #include "sphere.h"

// assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ float hit_sphere(const point3& center, float radius, const ray& r) {
    vec3 oc = center - r.origin();
    float a = r.direction().length_squared();
    float h = dot(r.direction(), oc);
    float c = oc.length_squared() - radius*radius;
    float discriminant = h*h - a*c;

    if (discriminant < 0) {
        return -1.0f;
    } else {
        return (h - std::sqrt(discriminant)) / a;
    }
}

__device__ color ray_color(const ray& r) {
    float t = hit_sphere(point3(0,0,-1), 0.5f, r);
    if (t > 0.0f) {
        vec3 N = unit_vector(r.at(t) - vec3(0,0,-1));
        return 0.5f*color(N.x()+1, N.y()+1, N.z()+1);
    }

    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0f, 1.0f, 1.0f) + a*color(0.5f, 0.7f, 1.0f);
}

// blue-white gradient from chapter 4
__global__ void render(vec3 *pixel_buffer, int img_width, int img_height,
    point3 pixel00_loc, vec3 pixel_delta_u, vec3 pixel_delta_v, point3 camera_center) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= img_width) || (j >= img_height)) return;

    point3 pixel_center = pixel00_loc+(i*pixel_delta_u)+(j*pixel_delta_v);
    vec3 ray_direction = pixel_center - camera_center;
    ray r(camera_center,ray_direction);
    pixel_buffer[j*img_width+i] = ray_color(r);
}

int main() {
    // select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // 1280 * 800 = 1,024,000 pixels, divisible by warp size 32
    // also divisible by thread block's row size (8)
    int img_width = 640, img_height = 360;      // match serial base
    // int img_width = 1280, img_height = 800;
    int num_pixels = img_width*img_height;

    // buffer to store device-calculated pixels, to later be printed on host;
    // using Unified Memory, i.e., accessible by both host and device
    // float *pixel_buffer;
    vec3 *pixel_buffer;
    CUDA_SAFE_CALL(hipMallocManaged((void **)&pixel_buffer, num_pixels*sizeof(vec3)));

    // square blocks to start
    int num_threads_per_block_row = 8;
    dim3 dimGrid(img_width/num_threads_per_block_row,
        img_height/num_threads_per_block_row);
    dim3 dimBlock(num_threads_per_block_row,num_threads_per_block_row);

    /* configure the virtual camera */
    float focal_length = 1.0f;
    float viewport_height = 2.0f;
    float viewport_width = viewport_height * (float(img_width)/img_height);
    point3 camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    vec3 viewport_u = vec3(viewport_width, 0, 0);
    vec3 viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    vec3 pixel_delta_u = viewport_u / img_width;
    vec3 pixel_delta_v = viewport_v / img_height;

    // Calculate the location of the upper left pixel.
    vec3 viewport_upper_left = camera_center
                             - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    vec3 pixel00_loc = viewport_upper_left + 0.5f * (pixel_delta_u + pixel_delta_v);
    /* end virtual camera config */

    // call the render() kernel
    render<<<dimGrid, dimBlock>>>(pixel_buffer, img_width, img_height,
        pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // output pixel_buffer as a .ppm image
    std::cout << "P3\n" << img_width << " " << img_height << "\n255\n";
    for (int j = 0; j < img_height; ++j) {      // rows
        for (int i = 0; i < img_width; ++i) {   // cols
            size_t pixel_index = j*img_width+i;
            int ir = int(255.99f*pixel_buffer[pixel_index].x());
            int ig = int(255.99f*pixel_buffer[pixel_index].y());
            int ib = int(255.99f*pixel_buffer[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    CUDA_SAFE_CALL(hipFree(pixel_buffer));

    return 0;
}
