#include "hip/hip_runtime.h"
// Note: Watch out! Include order will matter!
#include "rtweekend.h"
#include "hittable.h"
#include "color.h"
// #include "camera.h"
// #include "material.h"

// assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__device__ color ray_color(const ray& r, const world& world) {
    // track the hits for this particular ray
    hit_record rec;

    // hits will be the sphere's surface normal
    if (hit_world(world, r, interval(0.0f, infinity), rec)) {
        return 0.5f * (rec.normal + color(1,1,1));
    }

    // background blue-to-white gradient
    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0f, 1.0f, 1.0f) + a*color(0.5f, 0.7f, 1.0f);
}

__global__ void render(vec3 *pixel_buffer, int img_width, int img_height, point3 pixel00_loc,
    vec3 pixel_delta_u, vec3 pixel_delta_v, point3 camera_center,world *d_world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= img_width) || (j >= img_height)) return;

    point3 pixel_center = pixel00_loc+(i*pixel_delta_u)+(j*pixel_delta_v);
    vec3 ray_direction = pixel_center - camera_center;
    ray r(camera_center,ray_direction);
    pixel_buffer[j*img_width+i] = ray_color(r,*d_world);
}

__global__ void update_world_pointer(world *w, sphere *spheres) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        w->spheres = spheres;
    }
}

int main() {
    // select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // 1280 * 800 = 1,024,000 pixels, divisible by warp size 32
    // also divisible by thread block's row size (8)
    // these dimensions match serial base
    int img_width = 640, img_height = 360;
    // these dimensions match the CUDA reference
    // int img_width = 1280, img_height = 800;
    int num_pixels = img_width*img_height;

    // buffer to store device-calculated pixels, to later be printed on host;
    // using Unified Memory, i.e., accessible by both host and device
    vec3 *pixel_buffer;
    CUDA_SAFE_CALL(hipMallocManaged((void **)&pixel_buffer, num_pixels*sizeof(vec3)));

    // square blocks to start
    int num_threads_per_block_row = 8;
    dim3 dimGrid(img_width/num_threads_per_block_row,
        img_height/num_threads_per_block_row);
    dim3 dimBlock(num_threads_per_block_row,num_threads_per_block_row);

    /* virtual camera config */
    float focal_length = 1.0f;
    float viewport_height = 2.0f;
    float viewport_width = viewport_height * (float(img_width)/img_height);
    point3 camera_center = point3(0, 0, 0);

    // calculate the vectors across the horizontal and down the vertical viewport edges
    vec3 viewport_u = vec3(viewport_width, 0, 0);
    vec3 viewport_v = vec3(0, -viewport_height, 0);

    // calculate the horizontal and vertical delta vectors from pixel to pixel
    vec3 pixel_delta_u = viewport_u / img_width;
    vec3 pixel_delta_v = viewport_v / img_height;

    // calculate the location of the upper left pixel
    vec3 viewport_upper_left = camera_center
                             - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    vec3 pixel00_loc = viewport_upper_left + 0.5f * (pixel_delta_u + pixel_delta_v);
    /* end virtual camera config */

    /* world creation */
    // host allocations and initializations
    int num_spheres = 2;
    sphere *h_spheres = new sphere[num_spheres];
    h_spheres[0] = sphere(point3(0,0,-1), 0.5f);
    h_spheres[1] = sphere(point3(0,-100.5f,-1), 100);

    world *h_world = new world(h_spheres,num_spheres);

    // device allocations and transfers
    sphere *d_spheres;
    hipMalloc(&d_spheres, num_spheres*sizeof(sphere));
    hipMemcpy(d_spheres,h_spheres,num_spheres*sizeof(sphere),hipMemcpyHostToDevice);

    world *d_world;
    hipMalloc(&d_world,sizeof(world));
    hipMemcpy(d_world,h_world,sizeof(world),hipMemcpyHostToDevice);

    // update needed since the host pointer to spheres will be invalid
    // after transferring to device
    update_world_pointer<<<1,1>>>(d_world, d_spheres);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    /* end world creation*/

    // call the render() kernel
    render<<<dimGrid, dimBlock>>>(pixel_buffer, img_width, img_height, pixel00_loc,
        pixel_delta_u, pixel_delta_v, camera_center, d_world);
    CUDA_SAFE_CALL(hipGetLastError());
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // output pixel_buffer as a .ppm image
    std::cout << "P3\n" << img_width << " " << img_height << "\n255\n";
    for (int j = 0; j < img_height; ++j) {      // rows
        for (int i = 0; i < img_width; ++i) {   // cols
            size_t pixel_index = j*img_width+i;
            int ir = int(255.99f*pixel_buffer[pixel_index].x());
            int ig = int(255.99f*pixel_buffer[pixel_index].y());
            int ib = int(255.99f*pixel_buffer[pixel_index].z());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // hipFree and delete everything
    CUDA_SAFE_CALL(hipFree(pixel_buffer));
    delete h_spheres;
    delete h_world;

    return 0;
}
